#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>

int getSPcores(hipDeviceProp_t devProp)
{
//Note: this function was obtained from https://stackoverflow.com/questions/32530604/how-can-i-get-number-of-cores-in-cuda-device
	int cores = 0;
	int mp = devProp.multiProcessorCount;
	switch (devProp.major) {
	case 2: // Fermi
		if (devProp.minor == 1) cores = mp * 48;
		else cores = mp * 32;
		break;
	case 3: // Kepler
		cores = mp * 192;
		break;
	case 5: // Maxwell
		cores = mp * 128;
		break;
	case 6: // Pascal
		if ((devProp.minor == 1) || (devProp.minor == 2)) cores = mp * 128;
		else if (devProp.minor == 0) cores = mp * 64;
		else printf("Unknown device type\n");
		break;
	case 7: // Volta and Turing
		if ((devProp.minor == 0) || (devProp.minor == 5)) cores = mp * 64;
		else printf("Unknown device type\n");
		break;
	case 8: // Ampere
		if (devProp.minor == 0) cores = mp * 64;
		else if (devProp.minor == 6) cores = mp * 128;
		else if (devProp.minor == 9) cores = mp * 128; // ada lovelace
		else printf("Unknown device type\n");
		break;
	case 9: // Hopper
		if (devProp.minor == 0) cores = mp * 128;
		else printf("Unknown device type\n");
		break;
	default:
		printf("Unknown device type\n");
		break;
	}
	return cores;
}

int main(int argc, char *argv[]) {
	
	int nd;						//Variable to hold number of devices
	hipGetDeviceCount(&nd);
	for (int d = 0; d < nd; d++) {
		hipDeviceProp_t dp;					//Create a device properties object
		hipGetDeviceProperties(&dp, d);	//Gets device properties of device d and places it into dp
		printf("Device %d -->\n", d);
		printf("\tGPU Name: %s\n", dp.name);
		printf("\tGPU Clock Rate: %d KHz\n", dp.clockRate);
		printf("\tNumber of Mulitprocessors: %d\n", dp.multiProcessorCount);
		printf("\tNumber of CUDA Cores: %d\n", getSPcores(dp));
		printf("\tWarp Size: %d\n", dp.warpSize);
		printf("\tGlobal Memory: %d bytes\n", dp.totalGlobalMem);
		printf("\tConstant Memory: %d bytes\n", dp.totalConstMem);
		printf("\tShared Memory Per Block: %d bytes\n", dp.sharedMemPerBlock);
		printf("\tRegisters Per Block: %d\n", dp.regsPerBlock);
		printf("\tMax Threads Per Block: %d\n", dp.maxThreadsPerBlock);
		printf("\tMax Dimension of a Block: %d\n", dp.maxThreadsDim);
		printf("\tMax Dimension of a Grid: %d\n", dp.maxGridSize);
		//Refer to https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaDeviceProp.html
	}
}