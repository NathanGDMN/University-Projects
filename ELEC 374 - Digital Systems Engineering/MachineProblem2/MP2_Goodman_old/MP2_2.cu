#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

//2. Write a kernel that has each thread producing one output matrix. Kernel config should be 16x16 thread blocks

/* MatrixAddition Kernel.
Parameters: pointer to output matrix C, two Pointers to input matricies A and B,
dimensions of matricies A and B (remember they're square matriceies so this can be single int)
*/
__global__ void matrixAddition_kernel(float*d_a, float*d_b, float*d_c, int sizeOfMatricies) {
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	int idx = Row*sizeOfMatricies + Col;
	//0 1 2 3 4 
	//5 6 7 8 9
	if (idx < sizeOfMatricies*sizeOfMatricies) { //Avoid accessing beyond end off matricies
		d_c[idx] = d_a[idx] + d_b[idx];
	}
}


/* Created a matrixAddition function. Should basically be the same as the kernel function.
This function will be used to check whether the kernel function created correct output (used to check)
*/
void matrixAddition(float *a, float *b, float *c, int sizeOfMatricies) { //Note acc
	for (int i = 0; i < sizeOfMatricies; i++) {
		for (int j = 0; j < sizeOfMatricies; j++) {
			*(c + i*sizeOfMatricies + j) = *(a + i*sizeOfMatricies + j) + *(b + i*sizeOfMatricies + j);
			//C[i][j] = A[i][j] + B[i][j];
		}
	}
}

//3b. Create function to check if two matricies are equal (to be used to compare outputs)
int correct_output(float *a, float *b, int sizeOfMatricies) {
	for (int i = 0; i < sizeOfMatricies; i++) {
		for (int j = 0; j < sizeOfMatricies; j++) {
			if (*(a + i*sizeOfMatricies + j) != *(b + i*sizeOfMatricies + j)) {
				//If a[i][j] != b[i][j]:
				return 0;
			}
		}
	}
	return 1;
}

//Additional function to help with debugging:
void printMatrix(float *a, int size) {
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			printf("%f ", *(a + i*size + j));
		}
		printf("\n");
	}
}

//Function for generating  randomly intitialized square matricies of a given length
void fillMatrix(float *a, int size) {
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			*(a + i*size + j) = rand() % 100; //Every element will be a random number in range of 0 to 100
		}
	}
}

int main(int argc, char *argv[]) {
	hipDeviceProp_t deviceProps;

	//Get Device Name (They did this in tutorial so I'll do it here)
	hipGetDeviceProperties(&deviceProps, 0);
	printf("CUDA device [%s]\n", deviceProps.name);
	printf("\tNumber of Mulitprocessors: %d\n", deviceProps.multiProcessorCount);
	printf("\tMax Threads Per Block: %d\n", deviceProps.maxThreadsPerBlock);
	printf("\tMax Dimension of a Block: %d\n", deviceProps.maxThreadsDim);
	printf("\tMax Dimension of a Grid: %d\n", deviceProps.maxGridSize);

	/*
	1. Define two square input matricies A and B, and matching output matrix
	Note: they're floats
	*/
	const int dimOfMatricies = 250; //Value determines size of matricies Ex. dimOfMatricies = 5 will result in 5x5 matricies

	float C[dimOfMatricies][dimOfMatricies];

	//1b. Calculate amount of memory they take:
	int nbytes = dimOfMatricies*dimOfMatricies*sizeof(float);

	//1c. Allocate host memory for matricies:
	float *a = 0;
	float *b = 0;
	float *c = 0;
	hipHostMalloc((void**)&a, nbytes); //Allocates host memory for matrix A, and points pointer a to first value.
	hipHostMalloc((void**)&b, nbytes);
	hipHostMalloc((void**)&c, nbytes);

	//printf("a = %x\n",&a);
	//printf("b = %x\n", &b);
	//printf("c = %x\n", &c);

	//1d. Store input matricies into memory

	srand(time(NULL));
	fillMatrix(a, dimOfMatricies);
	fillMatrix(b, dimOfMatricies);

	//printf("%f\n", *b);
	//printf("Matrix A\n");
	//printMatrix(a, dimOfMatricies);
	//printf("Matrix B\n");
	//printMatrix(b, dimOfMatricies);

	//1e. Allocate device memory for matricies:
	float *d_a = 0;
	float *d_b = 0;
	float *d_c = 0;
	hipMalloc((void**)&d_a, nbytes); //Allocates memory for matrix A, and points pointer a to first value.
	hipMemset(d_a, 255, nbytes);	  //Sets all allocated bytes to 255 (they did this in tutorial so i did it here)
	hipMalloc((void**)&d_b, nbytes);
	hipMemset(d_b, 255, nbytes);
	hipMalloc((void**)&d_c, nbytes);
	hipMemset(d_c, 255, nbytes);

	//Set kernel launch configuration
	int blckWidth = 16; //block width and kength
	int threadsPerBlock = blckWidth*blckWidth;
	int threadsNeeded = dimOfMatricies*dimOfMatricies; //Because in this configuration one thread only produces one value
	int numBlocks = threadsNeeded / threadsPerBlock;
	if (numBlocks < 1) numBlocks++;

	dim3 dimBlocks = dim3(blckWidth, blckWidth);//asks for 16X16=256 thread blocks	
											//256 threads per block layed out in 16x16
	dim3 dimGrid = dim3(numBlocks, numBlocks);
								  
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipDeviceSynchronize();
	float gpu_time = 0.0f;

	//asynchronously issue work to the GPU (all stream 0)
	hipEventRecord(start, 0);

	//Copy inputs to device
	hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);
	hipMemcpyAsync(d_b, b, nbytes, hipMemcpyHostToDevice, 0);

	//Call Kernel
	matrixAddition_kernel << <dimGrid, dimBlocks, 0, 0 >> >(d_a, d_b, d_c, dimOfMatricies);
	//Note: only section after >>> is the actual function parameters

	//Copy outputs from device
	hipMemcpyAsync(c, d_c, nbytes, hipMemcpyDeviceToHost, 0);

	//2b. Load output matricies from memory
	for (int i = 0; i < dimOfMatricies; i++) {
		for (int j = 0; j < dimOfMatricies; j++) {
			C[i][j] = *(c + i*dimOfMatricies + j);
		}
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop); //stop is updated here
	hipEventElapsedTime(&gpu_time, start, stop);

	//print the GPU times
	printf("time spent executing by the GPU: %.2f\n", gpu_time);

	//Calculate matrixAddition using CPU:
	float D[dimOfMatricies][dimOfMatricies];
	float *d = 0;
	hipHostMalloc((void**)&d, nbytes);
	matrixAddition(a, b, d, dimOfMatricies); //d is output matrix

											 //Check Output
	bool bFinalResults = (bool)correct_output(c, d, dimOfMatricies); //check if c(from GPU) = d(from CPU)
	if (bFinalResults == true) {
		printf("Test PASSED\n");
	}

	//printf("Matrix C\n");
	//printMatrix(c, dimOfMatricies);
	//printf("\n");
	//printMatrix(d, dimOfMatricies);

	//release resources
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
	hipHostFree(d);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipDeviceReset();

	return 0;

}